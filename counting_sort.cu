
#include <hip/hip_runtime.h>
#include <cstdio>
#include <math.h> 
#include <ctime>
#include <iostream>
using namespace std;

int performanceMeasure1();
int performanceMeasure2();
int performanceMeasure3();
int performanceMeasure4();
int performanceMeasure5();
int countSortSerial1();
int countSortSerial2();
int countSortSerial3();
int countSortSerial4();
int countSortSerial5();

//calculate the countArray or histogram of number of times a key appears
__global__ void histogram(int * c, int * a, int K, int n)
{
//for inputArray of size n
	int entry =  (blockIdx.x + blockIdx.y * gridDim.x ) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	c[entry] = 0;
//if out of range then return
	if (entry < 0 || entry >= n) return;
//Get the value at the index
	int value = a[entry];
//update the counterArray at the value index by 1
	int *valueCount = &c[value];
	atomicAdd(valueCount, 1);
}

//calculate the prefix sum using a naive stride method
__global__ void naivePrefixSum(int *b, int *c, int k)
{

	int entry = threadIdx.x;
	if (entry < 0 || entry >= k) return;
	b[entry] = c[entry];
	//printf("c %d\n", b[entry]);	
	__syncthreads();
	//naive parallel stride prefix sum
	for(int i = 1; i < k; i *= 2)
	{
		if(entry > i-1) 
		{
			b[entry] = b[entry] + b[entry - i];
		}
		__syncthreads();
	}
	//printf("\nb %d", b[entry]); 
}

//from the prefix sum, place the numbers in the correct postion in the array
__global__ void copyToArray(int * c, int * a, int * b, int Kp, int n)
{
	extern __shared__ int temp[];
	int entry = (blockIdx.x + blockIdx.y * gridDim.x ) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	if (entry < 0 || entry >= n) return; 
   //get value at the inputArray at an index
	int value = a[entry];
   //get the index for the value 
	int index = atomicAdd(&c[value], -1);
	b[index-1] = value;
}


int main() {


//Start Debug Test
	printf("\nDebug Start\n");
///Test n elements with certain number of keys
	const int n = 1024;
	const int keys = 257;
//Setup Array on host and device
	int i_h[n] = {0};
	printf("\nInput:\n ");
	//An input array i_h (input array on host) with n elements with in the range of 0 to 256 and is a power of 2.
	for(int i = 0; i < n; i++){
		i_h[i] = pow(2,(std::rand() % 9));
		printf("%d ", i_h[i]);
	}
	int o_h[keys] = {0};
	int c_h[keys] = {0};
	int *i_d, *o_d, *c_d;
	//setup array on gpu
	hipMalloc((void **)&i_d, sizeof(int)*n);
	hipMalloc((void **)&o_d, sizeof(int)*n);
	hipMalloc((void **)&c_d, sizeof(int)*keys);
	//copy values from input,etc..
	hipMemcpy(i_d, i_h, sizeof(int)*n, hipMemcpyHostToDevice);
	hipMemcpy(o_d, o_d, sizeof(int)*n, hipMemcpyHostToDevice);
	hipMemcpy(c_d, c_h, sizeof(int)*keys, hipMemcpyHostToDevice);

//CountSortFunction
//Get histogram
	histogram <<<6, n>>>(c_d,i_d,keys,n);
	hipMemcpy(c_h, c_d, sizeof(int)*keys, hipMemcpyDeviceToHost);
//Calculate Prefix sum
	naivePrefixSum<<<1,n>>>(c_d,c_d,keys);
//Fill in array
	copyToArray<<<6,n>>>(c_d,i_d,o_d,keys,n);
//Get answer
	hipMemcpy(o_h, o_d, sizeof(int)*n, hipMemcpyDeviceToHost);
//print answer
	printf("\nOutput:\n ");
	for (int i = 0; i < n; ++i) printf("%d ", o_h[i]);
//free memory
		hipFree(i_d);
	hipFree(o_d);
	hipFree(c_d);
	
//Finish Debug Test
	printf("\nFinish debug\n");

//Performance test
	printf("Parallel function doesn't work on 2^21 and larger\n");
	printf("tried using clock but doesn't seem to work on serial function\n");
	printf("Debug test works");
	countSortSerial1();
	countSortSerial2();
	countSortSerial3();
	countSortSerial4();
	countSortSerial5();
	performanceMeasure1();
//performanceMeasure2();
//performanceMeasure3();
//performanceMeasure4();
//performanceMeasure5();
	
	return 0;
}

///////////////////////////////////////////////////////////Performance Function/////////////////////////////////////////////////////////////////
int countSortSerial1()
{   
	std::clock_t start;
	double duration;
	start = std::clock();
	const int elements = 1048576;
	const int keys = 257;

	int inputArray[elements] = {0};
	int output[elements] = {0};
	for(int i = 0; i < elements; i++)
		inputArray[i] = pow(2,(std::rand() % 9));

	int count[elements + 1] = {0};
	
    //Initalize the count array and count the number of keys
	for(int i = 0; inputArray[i]; ++i)
		++count[inputArray[i]];
	
    //calculate the starting index for each key
	int total = 0;
	int oldCount;
	for (int i = 0; i <= keys; ++i)
	{
		oldCount = count[i];
		count[i] = total;
		total += oldCount;
	}
	
    // Build the output character array
	for (int i = 0; inputArray[i]; ++i)
	{
		output[count[inputArray[i]]-1] = inputArray[i];
		--count[inputArray[i]];
	}
	
	cout << "serial counting 2^20 : " << duration << endl;
}

int countSortSerial2()
{   
	std::clock_t start;
	double duration;
	start = std::clock();
	const int elements = 1048576*2;
	const int keys = 257;

	int inputArray[elements] = {0};
	int output[elements] = {0};
	for(int i = 0; i < elements; i++)
		inputArray[i] = pow(2,(std::rand() % 9));

	int count[elements + 1] = {0};
	
    //Initalize the count array and count the number of keys
	for(int i = 0; inputArray[i]; ++i)
		++count[inputArray[i]];
	
    //calculate the starting index for each key
	int total = 0;
	int oldCount;
	for (int i = 0; i <= keys; ++i)
	{
		oldCount = count[i];
		count[i] = total;
		total += oldCount;
	}
	
    // Build the output character array
	for (int i = 0; inputArray[i]; ++i)
	{
		output[count[inputArray[i]]-1] = inputArray[i];
		--count[inputArray[i]];
	}
	
	cout << "serial counting 2^21 : " << duration << endl;
}

int countSortSerial3()
{   
	std::clock_t start;
	double duration;
	start = std::clock();
	const int elements = 1048576*2;
	const int keys = 257;

	int inputArray[elements] = {0};
	int output[elements] = {0};
	for(int i = 0; i < elements; i++)
		inputArray[i] = pow(2,(std::rand() % 9));

	int count[elements + 1] = {0};
	
    //Initalize the count array and count the number of keys
	for(int i = 0; inputArray[i]; ++i)
		++count[inputArray[i]];
	
    //calculate the starting index for each key
	int total = 0;
	int oldCount;
	for (int i = 0; i <= keys; ++i)
	{
		oldCount = count[i];
		count[i] = total;
		total += oldCount;
	}
	
    // Build the output character array
	for (int i = 0; inputArray[i]; ++i)
	{
		output[count[inputArray[i]]-1] = inputArray[i];
		--count[inputArray[i]];
	}
	
	cout << "serial counting 2^22 : " << duration << endl;
}

int countSortSerial4()
{   
	std::clock_t start;
	double duration;
	start = std::clock();
	const int elements = 1048576*2;
	const int keys = 257;

	int inputArray[elements] = {0};
	int output[elements] = {0};
	for(int i = 0; i < elements; i++)
		inputArray[i] = pow(2,(std::rand() % 9));

	int count[elements + 1] = {0};
	
    //Initalize the count array and count the number of keys
	for(int i = 0; inputArray[i]; ++i)
		++count[inputArray[i]];
	
    //calculate the starting index for each key
	int total = 0;
	int oldCount;
	for (int i = 0; i <= keys; ++i)
	{
		oldCount = count[i];
		count[i] = total;
		total += oldCount;
	}
	
    // Build the output character array
	for (int i = 0; inputArray[i]; ++i)
	{
		output[count[inputArray[i]]-1] = inputArray[i];
		--count[inputArray[i]];
	}
	
	cout << "serial counting 2^23 : " << duration << endl;
}

int countSortSerial5()
{   
	std::clock_t start;
	double duration;
	start = std::clock();
	const int elements = 1048576*2;
	const int keys = 257;

	int inputArray[elements] = {0};
	int output[elements] = {0};
	for(int i = 0; i < elements; i++)
		inputArray[i] = pow(2,(std::rand() % 9));

	int count[elements + 1] = {0};
	
    //Initalize the count array and count the number of keys
	for(int i = 0; inputArray[i]; ++i)
		++count[inputArray[i]];
	
    //calculate the starting index for each key
	int total = 0;
	int oldCount;
	for (int i = 0; i <= keys; ++i)
	{
		oldCount = count[i];
		count[i] = total;
		total += oldCount;
	}
	
    // Build the output character array
	for (int i = 0; inputArray[i]; ++i)
	{
		output[count[inputArray[i]]-1] = inputArray[i];
		--count[inputArray[i]];
	}
	
	cout << "serial counting 2^24 : " << duration << endl;
}
//Same function, but had trouble initalizing array from function parameter.
//so made copies of different performanceMeasure function 1 to 5 with different number of elements 2^20 to 2^24
int performanceMeasure1()
{
	std::clock_t start;
	double duration;
	start = std::clock();
//number of elements and number of keys
	const int elements = 1048576;
	const int keys = 257;
//setup device and host array variables
	int i_h[elements] = {0};
	for(int i = 0; i < elements; i++)
		i_h[i] = pow(2,(std::rand() % 9));
	int o_h[keys] = {0};
	int c_h[keys] = {0};
	int *i_d, *o_d, *c_d;
	//setup array on gpu
	hipMalloc((void **)&i_d, sizeof(int)*elements);
	hipMalloc((void **)&o_d, sizeof(int)*elements);
	hipMalloc((void **)&c_d, sizeof(int)*keys);
	//copy values from input,etc..
	hipMemcpy(i_d, i_h, sizeof(int)*elements, hipMemcpyHostToDevice);
	hipMemcpy(o_d, o_d, sizeof(int)*elements, hipMemcpyHostToDevice);
	hipMemcpy(c_d, c_h, sizeof(int)*keys, hipMemcpyHostToDevice);
//countsort
	//Get histogram
	histogram <<<6, elements>>>(c_d,i_d,keys,elements);
	hipMemcpy(c_h, c_d, sizeof(int)*keys, hipMemcpyDeviceToHost);
	//Calculate Prefix sum
	naivePrefixSum<<<1,elements>>>(c_d,c_d,keys);
	//Fill in array
	copyToArray<<<6,elements>>>(c_d,i_d,o_d,keys,elements);
	//Get answer
	hipMemcpy(o_h, o_d, sizeof(int)*elements, hipMemcpyDeviceToHost);
	//free memory
	hipFree(i_d);
	hipFree(o_d);
	hipFree(c_d);
	duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
	cout << "parallel counting 2^20 : " << duration << endl;
	return 0;
}


int performanceMeasure2()
{
	std::clock_t start;
	double duration;
	start = std::clock();
//number of elements and number of keys
	const int elements = 2097152;
	const int keys = 257;
//setup device and host array variables
	int i_h[elements] = {0};
	for(int i = 0; i < elements; i++)
		i_h[i] = pow(2,(std::rand() % 9));
	int o_h[keys] = {0};
	int c_h[keys] = {0};
	int *i_d, *o_d, *c_d;
	//setup array on gpu
	hipMalloc((void **)&i_d, sizeof(int)*elements);
	hipMalloc((void **)&o_d, sizeof(int)*elements);
	hipMalloc((void **)&c_d, sizeof(int)*keys);
	//copy values from input,etc..
	hipMemcpy(i_d, i_h, sizeof(int)*elements, hipMemcpyHostToDevice);
	hipMemcpy(o_d, o_d, sizeof(int)*elements, hipMemcpyHostToDevice);
	hipMemcpy(c_d, c_h, sizeof(int)*keys, hipMemcpyHostToDevice);
//countsort
	//Get histogram
	histogram <<<6, elements>>>(c_d,i_d,keys,elements);
	hipMemcpy(c_h, c_d, sizeof(int)*keys, hipMemcpyDeviceToHost);
	//Calculate Prefix sum
	naivePrefixSum<<<1,elements>>>(c_d,c_d,keys);
	//Fill in array
	copyToArray<<<6,elements>>>(c_d,i_d,o_d,keys,elements);
	//Get answer
	hipMemcpy(o_h, o_d, sizeof(int)*elements, hipMemcpyDeviceToHost);
	//free memory
	hipFree(i_d);
	hipFree(o_d);
	hipFree(c_d);
	cout << "parallel counting 2^21 : " << duration << endl;
	return 0;
}


int performanceMeasure3()
{
	std::clock_t start;
	double duration;
	start = std::clock();
//number of elements and number of keys
	const int elements = 4194304;
	const int keys = 257;
//setup device and host array variables
	int i_h[elements] = {0};
	for(int i = 0; i < elements; i++)
		i_h[i] = pow(2,(std::rand() % 9));
	int o_h[keys] = {0};
	int c_h[keys] = {0};
	int *i_d, *o_d, *c_d;
	//setup array on gpu
	hipMalloc((void **)&i_d, sizeof(int)*elements);
	hipMalloc((void **)&o_d, sizeof(int)*elements);
	hipMalloc((void **)&c_d, sizeof(int)*keys);
	//copy values from input,etc..
	hipMemcpy(i_d, i_h, sizeof(int)*elements, hipMemcpyHostToDevice);
	hipMemcpy(o_d, o_d, sizeof(int)*elements, hipMemcpyHostToDevice);
	hipMemcpy(c_d, c_h, sizeof(int)*keys, hipMemcpyHostToDevice);
//countsort
	//Get histogram
	histogram <<<6, elements>>>(c_d,i_d,keys,elements);
	hipMemcpy(c_h, c_d, sizeof(int)*keys, hipMemcpyDeviceToHost);
	//Calculate Prefix sum
	naivePrefixSum<<<1,elements>>>(c_d,c_d,keys);
	//Fill in array
	copyToArray<<<6,elements>>>(c_d,i_d,o_d,keys,elements);
	//Get answer
	hipMemcpy(o_h, o_d, sizeof(int)*elements, hipMemcpyDeviceToHost);
	//free memory
	hipFree(i_d);
	hipFree(o_d);
	hipFree(c_d);
	cout << "parallel counting 2^22 : " << duration << endl;
	return 0;
}


int performanceMeasure4()
{
	std::clock_t start;
	double duration;
	start = std::clock();
//number of elements and number of keys
	const int elements = 8388608;
	const int keys = 257;
//setup device and host array variables
	int i_h[elements] = {0};
	for(int i = 0; i < elements; i++)
		i_h[i] = pow(2,(std::rand() % 9));
	int o_h[keys] = {0};
	int c_h[keys] = {0};
	int *i_d, *o_d, *c_d;
	//setup array on gpu
	hipMalloc((void **)&i_d, sizeof(int)*elements);
	hipMalloc((void **)&o_d, sizeof(int)*elements);
	hipMalloc((void **)&c_d, sizeof(int)*keys);
	//copy values from input,etc..
	hipMemcpy(i_d, i_h, sizeof(int)*elements, hipMemcpyHostToDevice);
	hipMemcpy(o_d, o_d, sizeof(int)*elements, hipMemcpyHostToDevice);
	hipMemcpy(c_d, c_h, sizeof(int)*keys, hipMemcpyHostToDevice);
//countsort
	//Get histogram
	histogram <<<6, elements>>>(c_d,i_d,keys,elements);
	hipMemcpy(c_h, c_d, sizeof(int)*keys, hipMemcpyDeviceToHost);
	//Calculate Prefix sum
	naivePrefixSum<<<1,elements>>>(c_d,c_d,keys);
	//Fill in array
	copyToArray<<<6,elements>>>(c_d,i_d,o_d,keys,elements);
	//Get answer
	hipMemcpy(o_h, o_d, sizeof(int)*elements, hipMemcpyDeviceToHost);
	//free memory
	hipFree(i_d);
	hipFree(o_d);
	hipFree(c_d);
	cout << "parallel counting 2^23 : " << duration << endl;
	return 0;
}


int performanceMeasure5()
{
	std::clock_t start;
	double duration;
	start = std::clock();
//number of elements and number of keys
	const int elements = 16777216;
	const int keys = 257;
//setup device and host array variables
	int i_h[elements] = {0};
	for(int i = 0; i < elements; i++)
		i_h[i] = pow(2,(std::rand() % 9));
	int o_h[keys] = {0};
	int c_h[keys] = {0};
	int *i_d, *o_d, *c_d;
	//setup array on gpu
	hipMalloc((void **)&i_d, sizeof(int)*elements);
	hipMalloc((void **)&o_d, sizeof(int)*elements);
	hipMalloc((void **)&c_d, sizeof(int)*keys);
	//copy values from input,etc..
	hipMemcpy(i_d, i_h, sizeof(int)*elements, hipMemcpyHostToDevice);
	hipMemcpy(o_d, o_d, sizeof(int)*elements, hipMemcpyHostToDevice);
	hipMemcpy(c_d, c_h, sizeof(int)*keys, hipMemcpyHostToDevice);
//countsort
	//Get histogram
	histogram <<<6, elements>>>(c_d,i_d,keys,elements);
	hipMemcpy(c_h, c_d, sizeof(int)*keys, hipMemcpyDeviceToHost);
	//Calculate Prefix sum
	naivePrefixSum<<<1,elements>>>(c_d,c_d,keys);
	//Fill in array
	copyToArray<<<6,elements>>>(c_d,i_d,o_d,keys,elements);
	//Get answer
	hipMemcpy(o_h, o_d, sizeof(int)*elements, hipMemcpyDeviceToHost);
	//free memory
	hipFree(i_d);
	hipFree(o_d);
	hipFree(c_d);
	cout << "parallel counting 2^24 : " << duration << endl;
	return 0;
}
